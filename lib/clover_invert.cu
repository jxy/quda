#include "hip/hip_runtime.h"
#include <clover_field.h>
#include <instantiate.h>
#include <tunable_reduction.h>
#include <kernels/clover_invert.cuh>

namespace quda {

  template <typename store_t>
  class CloverInvert : TunableReduction2D<> {
    CloverField &clover;
    bool compute_tr_log;

  public:
    CloverInvert(CloverField &clover, bool compute_tr_log) :
      TunableReduction2D(clover),
      clover(clover),
      compute_tr_log(compute_tr_log)
    {
      writeAuxString("trlog=%s,twist=%s",
                     compute_tr_log ? "true" : "false",
                     clover.Twisted() ? "true" : "false");

      apply(device::get_default_stream());

      if (compute_tr_log && (std::isnan(clover.TrLog()[0]) || std::isnan(clover.TrLog()[1]))) {
	printfQuda("clover.TrLog()[0]=%e, clover.TrLog()[1]=%e\n", clover.TrLog()[0], clover.TrLog()[1]);
	errorQuda("Clover trlog has returned -nan, likey due to the clover matrix being singular.");
      }
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (clover.Twisted()) {
        CloverInvertArg<store_t, true> arg(clover, compute_tr_log);
        launch<InvertClover>(clover.TrLog(), tp, stream, arg);
      } else {
        CloverInvertArg<store_t, false> arg(clover, compute_tr_log);
        launch<InvertClover>(clover.TrLog(), tp, stream, arg);
      }
    }
    
    long long flops() const { return 0; }
    long long bytes() const { return 2 * clover.Bytes(); }
    void preTune() { if (clover.V(true) == clover.V(false)) clover.backup(); }
    void postTune() { if (clover.V(true) == clover.V(false)) clover.restore(); }
  };

#ifdef GPU_CLOVER_DIRAC
  void cloverInvert(CloverField &clover, bool computeTraceLog)
  {
    instantiate<CloverInvert>(clover, computeTraceLog);
  }
#else
  void cloverInvert(CloverField &, bool)
  {
    errorQuda("Clover has not been built");
  }
#endif

} // namespace quda
